#include "hip/hip_runtime.h"
#include "gpu_desm_utils.h"
#include "hip/hip_runtime.h"
#include "stdio.h"

__global__
void KERNEL_EW_FloatMultiVecAdd(const float * ptr_dict_d_,
                                const int * title_word_matrix,
                                const int * title_sent_len,
                                const int batch_size,
                                const int max_len,
                                const int vec_dim,
                                float * ptr_out_d_){




    float *local_ptr_d_out = ptr_out_d_ + blockIdx.x*vec_dim;

    for(int j=0; j<title_sent_len[blockIdx.x]; j++) {

        int word_offset = title_word_matrix[blockIdx.x*max_len+j];

        const float * word_vec_start = ptr_dict_d_ + word_offset*vec_dim;

        for(int i=threadIdx.x; i<vec_dim; i+=blockDim.x){
            local_ptr_d_out[i] += word_vec_start[i];
        }
    }
 }


void EW_FloatMultiVecAdd( const float * ptr_dict_d_,
                          const int * title_word_matrix,
                          const int * title_sent_len,
                          const int batch_size,
                          const int max_len,
                          const int vec_dim,
                          float * ptr_out_d_)
{

    int block_size  = batch_size;
    int thread_size = 32;

    hipMemset(ptr_out_d_, 0,sizeof(float)*batch_size*vec_dim);

    KERNEL_EW_FloatMultiVecAdd <<< block_size, thread_size >>> (
            ptr_dict_d_,
            title_word_matrix,
            title_sent_len,
            batch_size
            max_len,
            vec_dim,
            ptr_out_d_);

#if 0
    float *tmp = NULL;



    tmp = static_cast<float *> (malloc(sizeof(float)*batch_size*100));

    hipMemcpy(tmp,ptr_d_out_start,sizeof(float)*batch_size*100,hipMemcpyDeviceToHost);
    for(int i=0; i< batch_size; ++i) {
        for(int k =0; k<100; ++k) {
            printf("%g ",tmp[i*100+k]);

        }
        printf("\n");
                printf("\n");

        printf("\n");


    }
#endif


}
